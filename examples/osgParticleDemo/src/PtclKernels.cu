
#include <hip/hip_runtime.h>
/* osgCompute - Copyright (C) 2008-2009 SVT Group
*                                                                     
* This library is free software; you can redistribute it and/or modify
* it under the terms of the GNU Lesser General Public License as
* published by the Free Software Foundation; either version 3 of
* the License, or (at your option) any later version.
*                                                                     
* This library is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of 
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU Lesse General Public License for more details.
*
* The full license is in LICENSE file included with this distribution.
*/

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
inline __device__ 
float lerp(float a, float b, float t)
{
    return a + t*(b-a);
}

//------------------------------------------------------------------------------
inline __device__ 
float4 operator+(float4 a, float4 b)
{
    return make_float4(a.x + b.x, a.y + b.y, a.z + b.z,  a.w + b.w); 
}


//------------------------------------------------------------------------------
inline __device__
float4 seed( float* seeds, unsigned int seedCount, unsigned int seedIdx, unsigned int ptclIdx, float3 bbmin, float3 bbmax )
{
    // random seed idx
    unsigned int idx1 = (seedIdx + ptclIdx) % seedCount;
    unsigned int idx2 = (idx1 + ptclIdx) % seedCount;
    unsigned int idx3 = (idx2 + ptclIdx) % seedCount;

    // seeds are within the range [0,1]
    float intFac1 = seeds[idx1];
    float intFac2 = seeds[idx2];
    float intFac3 = seeds[idx3];

    return make_float4(lerp(bbmin.x,bbmax.x,intFac1), lerp(bbmin.y,bbmax.y,intFac3),
        lerp(bbmin.z,bbmax.z,intFac2), 1);
}

//------------------------------------------------------------------------------
inline __device__
unsigned int thIdx()
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int width = gridDim.x * blockDim.x;

    return y*width + x;
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GLOBAL FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
__global__
void reseedKernel( float4* ptcls, 
                   float* seeds, 
                   unsigned int seedCount, 
                   unsigned int seedIdx, 
                   float3 bbmin, 
                   float3 bbmax, 
                   unsigned int numPtcls )
{
    // Receive particle pos
    unsigned int ptclIdx = thIdx();
    if( ptclIdx < numPtcls )
    {
        float4 curPtcl = ptcls[ptclIdx];

        // Reseed Particles if they
        // have moved out of the bounding box
        if( curPtcl.x < bbmin.x ||
            curPtcl.y < bbmin.y ||
            curPtcl.z < bbmin.z ||
            curPtcl.x > bbmax.x ||
            curPtcl.y > bbmax.y ||
            curPtcl.z > bbmax.z )
            ptcls[ptclIdx] = seed( seeds, seedCount, seedIdx, ptclIdx, bbmin, bbmax );
    }
}

//------------------------------------------------------------------------------
__global__
void moveKernel( float4* ptcls, 
                 float etime, 
                 unsigned int numPtcls )
{
    unsigned int ptclIdx = thIdx();
    if( ptclIdx < numPtcls )
    {
        // perform a euler step
        ptcls[ptclIdx] = ptcls[ptclIdx] + make_float4(0,0,etime,0);
    }
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS ////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
extern "C" __host__
void reseed(unsigned int numBlocks, 
            unsigned int numThreads, 
            void* ptcls, 
            void* seeds, 
            unsigned int seedCount, 
            unsigned int seedIdx, 
            float3 bbmin, 
            float3 bbmax,
            unsigned int numPtcls)
{
    dim3 blocks( numBlocks, 1, 1 );
    dim3 threads( numThreads, 1, 1 );


    reseedKernel<<< blocks, threads >>>(
        (float4*)ptcls,
        (float*)seeds,
        seedCount,
        seedIdx,
        bbmin,
        bbmax,
        numPtcls );
}

//------------------------------------------------------------------------------
extern "C" __host__
void move( unsigned int numBlocks, 
           unsigned int numThreads, 
           void* ptcls, 
           float etime,
           unsigned int numPtcls )
{
    dim3 blocks( numBlocks, 1, 1 );
    dim3 threads( numThreads, 1, 1 );

    moveKernel<<< blocks, threads >>>( 
        (float4*)ptcls,
        etime,
        numPtcls );
}